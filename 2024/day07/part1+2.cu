
#include <hip/hip_runtime.h>
#include <atomic>
#include <fstream>
#include <iostream>
#include <sstream>
#include <thread>
#include <vector>

#ifndef PART2
  #define __A 2
  #define __B 1
  #define __C 1
#else
  #define __A 4
  #define __B 2
  #define __C 3
#endif

__device__ unsigned long concat(unsigned long a, unsigned long b) {
  unsigned long mp = 1;
  while (mp <= b) mp *= 10;
  return a * mp + b;
}

__global__ void find_operators(
  // IN
  const unsigned int    total_combinations,
  const unsigned int  * operands,
  const unsigned int    operands_count,
  const unsigned long   target,

  // OUT
  unsigned int  * found
) {
  unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;

  ///// ///// ///// ///// /////

  if (idx >= total_combinations || *found) {
    return;
  }

  ///// ///// ///// ///// /////

  unsigned long result = operands[0];

  for (unsigned int i = 1; i < operands_count; i++) {
    switch ((idx >> ((i - 1) * __B)) & __C) {
      case 0: result += operands[i]; break;
      case 1: result *= operands[i]; break;
      case 2: result = concat(result, operands[i]); break;
      default: return;
    }
  }

  ///// ///// ///// ///// /////

  if (result == target) {
    atomicExch(found, 1);
  }
}

int main() {
  std::ifstream file("input.txt");
  std::string line;
  std::vector<std::thread> threads;
  std::atomic<unsigned long> results = 0;

  while (std::getline(file, line)) {
    std::istringstream stream(line);

    ///// ///// ///// ///// /////

    unsigned long target;
    unsigned char colon;
    unsigned int operand;

    std::vector<unsigned int> operands_vec;

    ///// ///// ///// ///// /////

    if (stream >> target) {
      stream >> colon;

      while (stream >> operand) {
        operands_vec.push_back(operand);
      }
    }

    ///// ///// ///// ///// /////

    threads.push_back(
      std::thread(
        [& results, target, operands_vec] () {
          const unsigned int  * h_operands       = operands_vec.data();
          const unsigned int    h_operands_count = operands_vec.size();
          const unsigned long   h_target         = target;
                unsigned int    h_found          = 0;

                unsigned int  * d_operands;
                unsigned int  * d_found;

          ///// ///// ///// ///// /////

          hipMalloc(&d_operands, h_operands_count * sizeof(unsigned int));
          hipMalloc(&d_found   ,                    sizeof(unsigned int));

          ///// ///// ///// ///// /////

          hipMemcpy( d_operands,  h_operands, h_operands_count * sizeof(unsigned int), hipMemcpyHostToDevice);
          hipMemcpy( d_found   , &h_found   ,                    sizeof(unsigned int), hipMemcpyHostToDevice);

          ///// ///// ///// ///// /////

          unsigned int threads_per_block = 256;
          unsigned int total_combinations = pow(__A, h_operands_count - 1);
          unsigned int blocks = (total_combinations + threads_per_block - 1) / threads_per_block;
          find_operators<<<blocks, threads_per_block>>>(total_combinations, d_operands, h_operands_count, h_target, d_found);

          ///// ///// ///// ///// /////

          hipMemcpy(&h_found   ,  d_found   ,                    sizeof(unsigned int), hipMemcpyDeviceToHost);

          ///// ///// ///// ///// /////

          if (h_found) {
            results += h_target;
          }
        }
      )
    );
  }

  ///// ///// ///// ///// /////

  for (auto & thread : threads) {
    thread.join();
  }

  ///// ///// ///// ///// /////

  std::cout << results << std::endl;

  ///// ///// ///// ///// /////

  file.close();

  ///// ///// ///// ///// /////

  return 0;
}
